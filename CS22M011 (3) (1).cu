#include <iostream>
#include <stdio.h>
#include <hip/hip_runtime.h>

#define max_N 100000
#define max_P 30
#define BLOCKSIZE 1024

using namespace std;

//*******************************************

__global__ void AssignRequests(int *CentreRequests,int *RequestFrequency,int *facility,int *req_start,int *req_slots,int *capacity,int *total, int *success,int *tot,int *suc,int R,int N)
{
  int id=blockDim.x*blockIdx.x+threadIdx.x; 
  if(id<N*max_P)
  {
    int id1=id/max_P,id2=id%max_P; // id1 resembles the centre id whereas id2 resembles the facility within that centre
    if(id2<facility[id1])  // Facility id should be less than the number of facilities in that centre
    {
      int cap=capacity[id1*max_P+id2],count=0,start,end,i,j; // Capacity the facility
      int assign[24]; // This array is used to assign slots to requests. It's indexe's value tells how many requests are running on that slot.
      for(i=0;i<24;i++)
      {
        assign[i]=0;
      }
      for(i=0;i<RequestFrequency[id1*max_P+id2];i++) // A loop that iterates over all the requests on a particular facility.
      {
        start=req_start[CentreRequests[i+id1*max_P*R+id2*R]]-1; // Starting slot needed by a request
        end=start+req_slots[CentreRequests[i+id1*max_P*R+id2*R]];  // End slot needed by a request
        for(j=start;j<end;j++)
        {
          if(assign[j]>=cap) // It means if any slot is at it's full capacity
          {
            break;
          }
        }
        if(j==end) // It means no slot in the range start to end is on it's full capacity. Hence, the request can be fullfilled.
        {
          for(j=start;j<end;j++)
          {
            assign[j]++; // Increasing all slots by 1 cause they are running a new request.
          }
          count++; // Stores the number of successful requests.
        }
      }
      atomicAdd(&total[id1],RequestFrequency[id1*max_P+id2]); // Stores the total number of requests per center
      atomicAdd(&success[id1],count); // Stores successful requests per center
      atomicAdd(tot,RequestFrequency[id1*max_P+id2]); // Stores the total number of requests overall
      atomicAdd(suc,count); // Stores successful requests overall
    }
  }
}
//***********************************************


int main(int argc,char **argv)
{
	// variable declarations...
    int N,*centre,*facility,*capacity,*fac_ids, *succ_reqs, *tot_reqs;
    

    FILE *inputfilepointer;
    
    //File Opening for read
    char *inputfilename = argv[1];
    inputfilepointer    = fopen( inputfilename , "r");

    if ( inputfilepointer == NULL )  {
        printf( "input.txt file failed to open." );
        return 0; 
    }

    fscanf( inputfilepointer, "%d", &N ); // N is number of centres
	
    // Allocate memory on cpu
    centre=(int*)malloc(N * sizeof (int));  // Computer  centre numbers
    facility=(int*)malloc(N * sizeof (int));  // Number of facilities in each computer centre
    fac_ids=(int*)malloc(max_P * N  * sizeof (int));  // Facility room numbers of each computer centre
    capacity=(int*)malloc(max_P * N * sizeof (int));  // stores capacities of each facility for every computer centre 


    int success=0;  // total successful requests
    int fail = 0;   // total failed requests
    tot_reqs = (int *)malloc(N*sizeof(int));   // total requests for each centre
    succ_reqs = (int *)malloc(N*sizeof(int)); // total successful requests for each centre

    // Input the computer centres data
    for(int i=0;i<N;i++)
    {
      fscanf( inputfilepointer, "%d", &centre[i] );
      fscanf( inputfilepointer, "%d", &facility[i] );
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &fac_ids[i*max_P+j] );
      }
      for(int j=0;j<facility[i];j++)
      {
        fscanf( inputfilepointer, "%d", &capacity[i*max_P+j]);   
      }
    }

    // variable declarations
    int *req_id, *req_cen, *req_fac, *req_start, *req_slots;   // Number of slots requested for every request
    
    // Allocate memory on CPU 
	int R;
	fscanf( inputfilepointer, "%d", &R); // Total requests
    req_id = (int *) malloc ( (R) * sizeof (int) );  // Request ids
    req_cen = (int *) malloc ( (R) * sizeof (int) );  // Requested computer centre
    req_fac = (int *) malloc ( (R) * sizeof (int) );  // Requested facility
    req_start = (int *) malloc ( (R) * sizeof (int) );  // Start slot of every request
    req_slots = (int *) malloc ( (R) * sizeof (int) );   // Number of slots requested for every request
    
    // Input the user request data
    for(int j = 0; j < R; j++)
    {
       fscanf( inputfilepointer, "%d", &req_id[j]);
       fscanf( inputfilepointer, "%d", &req_cen[j]);
       fscanf( inputfilepointer, "%d", &req_fac[j]);
       fscanf( inputfilepointer, "%d", &req_start[j]);
       fscanf( inputfilepointer, "%d", &req_slots[j]);
       tot_reqs[req_cen[j]]+=1;  
    }
		

    int *d_req_id,*d_facility,*d_total,*d_success,*d_capacity,*d_t,*d_s,*d_req_cen, *d_req_fac, *d_req_start, *d_req_slots,*CentreRequests,*RequestFrequency,*d_CentreRequests,*d_RequestFrequency;
    RequestFrequency=(int*)malloc((N *max_P)* sizeof (int));
    memset(RequestFrequency, 0, (N*max_P)*sizeof(int));

    hipMalloc(&d_t, sizeof(int));  // Variable to store total number of requests
    hipMalloc(&d_s, sizeof(int));  // Variable to store total successful requests
    hipMalloc(&d_total, (N)*sizeof(int));  // Variable to store total requests per centre
    hipMalloc(&d_success, (N)*sizeof(int));  // Variable to store successful requests per centre
    hipMalloc(&d_facility, (N)*sizeof(int));
    hipMalloc(&d_req_id, (R)*sizeof(int));
    hipMalloc(&d_req_cen, (R)*sizeof(int));
    hipMalloc(&d_req_fac, (R)*sizeof(int));
    hipMalloc(&d_req_start, (R)*sizeof(int));
    hipMalloc(&d_req_slots, (R)*sizeof(int));
    hipMalloc(&d_RequestFrequency, (N*max_P)*sizeof(int));  // Variable to store the number of requests per facility
    hipMalloc(&d_capacity, (max_P*N)*sizeof(int));

    hipMemcpy(d_facility, facility, (N)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_id, req_id, (R)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_cen, req_cen, (R)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_fac, req_fac, (R)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_start, req_start, (R)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_req_slots,req_slots, (R)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_capacity,capacity, (N*max_P)*sizeof(int), hipMemcpyHostToDevice);
    hipMemset(&d_total, 0, N*sizeof(int));
    hipMemset(&d_success, 0, N*sizeof(int));
    int ma=0; // Variable to store maximum number of requests at a particluar facility for given input
    for(int i=0;i<R;i++)
    {
      RequestFrequency[req_cen[i]*max_P+req_fac[i]]++;  // Incrementing the frequency of requests in a particular facility 
      if(ma<RequestFrequency[req_cen[i]*max_P+req_fac[i]])
      {
        ma=RequestFrequency[req_cen[i]*max_P+req_fac[i]];  // Updating maximum
      }
    }
    memset(RequestFrequency, 0, (N*max_P)*sizeof(int)); // Initializing it to all 0's cause we need to use it again.
    CentreRequests=(int*)malloc((N *max_P * ma)* sizeof (int)); // A variable used to store request id's of requests per facility.
    hipMalloc(&d_CentreRequests, (ma * N * max_P)*sizeof(int));
    for(int i=0;i<R;i++)
    {
      // Here since we are iterating serially and requests id's are 0 to R-1, they would already be sorted. Hence, no sorting needed.
      CentreRequests[req_cen[i]*ma*max_P+req_fac[i]*ma+RequestFrequency[req_cen[i]*max_P+req_fac[i]]]=req_id[i];// Storing requests id's corresponding to the facility they are incident on.
      RequestFrequency[req_cen[i]*max_P+req_fac[i]]++;// It now resemebles the next index in CentreRequests where id's need to be added.
    }
    hipMemcpy(d_CentreRequests,CentreRequests, N*max_P*ma*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_RequestFrequency,RequestFrequency, (N*max_P)*sizeof(int), hipMemcpyHostToDevice);
    // This kernel basically runs all facility rooms in parallel and assign the requests that they can execute.
    AssignRequests<<<ceil((float)(N*max_P)/1024.0),1024>>>(d_CentreRequests,d_RequestFrequency,d_facility,d_req_start,d_req_slots,d_capacity,d_total,d_success,d_t,d_s,ma,N);
    hipMemcpy(tot_reqs,d_total,N*sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(succ_reqs,d_success,N*sizeof(int),hipMemcpyDeviceToHost);
    int *t,*s;
    t=(int*)malloc(sizeof (int)); // Variable to stores total requests in CPU
    s=(int*)malloc(sizeof (int));  // Variable to stores successful requests in CPU
    hipMemcpy(s,d_s,sizeof(int),hipMemcpyDeviceToHost);
    hipMemcpy(t,d_t,sizeof(int),hipMemcpyDeviceToHost);
    success=*s;
    fail=*t-success;// Failed requets = Total requests - Successful requests
    //********************************

    // Output
    char *outputfilename = argv[2]; 
    FILE *outputfilepointer;
    outputfilepointer = fopen(outputfilename,"w");

    fprintf( outputfilepointer, "%d %d\n", success, fail);
    for(int j = 0; j < N; j++)
    {
        fprintf( outputfilepointer, "%d %d\n", succ_reqs[j], tot_reqs[j]-succ_reqs[j]);
    }
    fclose( inputfilepointer );
    fclose( outputfilepointer );
    hipDeviceSynchronize();
	return 0;
}